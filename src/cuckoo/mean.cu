#include "hip/hip_runtime.h"
/** Cuckoo Cycle, a memory-hard proof-of-work by John Tromp
 * Copyright (c) 2018 Jiri Vadura (photon) and John Tromp
 * Copyright (c) 2018 Miguel Padilla
 *
 * This software is covered by the FAIR MINING license
 */

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include <chrono>


#ifdef _WIN32
#include "../windows/getopt.h"
#else
#include <unistd.h>
#endif

#include "cuckoo.h"
#include "../crypto/siphash.cuh"
#include "../crypto/blake2.h"
#include "../crypto/base64.h"

bool will_debug = false;

typedef uint32_t node_t;
typedef uint64_t nonce_t;

typedef std::chrono::milliseconds ms;

#ifndef XBITS
#define XBITS ((EDGEBITS-16)/2)
#endif

#define NODEBITS (EDGEBITS + 1)
#define NNODES ((node_t)1 << NODEBITS)
#define NODEMASK (NNODES - 1)

const static uint32_t NX        = 1 << XBITS;
const static uint32_t NX2       = NX * NX;
const static uint32_t XMASK     = NX - 1;
const static uint32_t X2MASK    = NX2 - 1;
const static uint32_t YBITS     = XBITS;
const static uint32_t NY        = 1 << YBITS;
const static uint32_t YZBITS    = EDGEBITS - XBITS;
const static uint32_t NYZ       = 1 << YZBITS;
const static uint32_t ZBITS     = YZBITS - YBITS;
const static uint32_t NZ        = 1 << ZBITS;

#define EPS_A 133/128
#define EPS_B 85/128

const static uint32_t ROW_EDGES_A = NYZ * EPS_A;
const static uint32_t ROW_EDGES_B = NYZ * EPS_B;

const static uint32_t EDGES_A = ROW_EDGES_A / NX;
const static uint32_t EDGES_B = ROW_EDGES_B / NX;
int global_device_id = 0;

__constant__ uint2 recoveredges[PROOFSIZE];
__constant__ uint2 e0 = {0,0};

__device__ __forceinline__ uint4 Pack8(const uint32_t e0, const uint32_t e1, const uint32_t e2, const uint32_t e3, const uint32_t e4, const uint32_t e5, const uint32_t e6, const uint32_t e7)
{
  return make_uint4((uint64_t)e0<<32|e1, (uint64_t)e2<<32|e3, (uint64_t)e4<<32|e5, (uint64_t)e6<<32|e7);
}

#ifndef FLUSHA // should perhaps be in trimparams and passed as template parameter
#define FLUSHA 16
#endif

template<int maxOut, typename EdgeOut>
__global__ void SeedA(const siphash_keys &sipkeys, uint4 * __restrict__ buffer, int * __restrict__ indexes)
{
    const int group = blockIdx.x;
    const int dim = blockDim.x;
    const int lid = threadIdx.x;
    const int gid = group * dim + lid;
    const int nthreads = gridDim.x * dim;
    const int FLUSHA2 = 2*FLUSHA;

    __shared__ EdgeOut tmp[NX][FLUSHA2]; // needs to be uint4 aligned
    const int TMPPERLL4 = sizeof(uint4) / sizeof(EdgeOut);
    __shared__ int counters[NX];

    for (int row = lid; row < NX; row += dim)
    {
        counters[row] = 0;
    }
    __syncthreads();

    const int col = group % NX;
    const int loops = NEDGES / nthreads;

    for (int i = 0; i < loops; i++)
    {
      uint32_t nonce = gid * loops + i;
      uint32_t node1, node0 = dipnode(sipkeys, (uint64_t)nonce, 0);

      if (sizeof(EdgeOut) == sizeof(uint2))
      {
          node1 = dipnode(sipkeys, (uint64_t)nonce, 1);
      }

      int row = node0 & XMASK;
      int counter = min((int)atomicAdd(counters + row, 1), (int)(FLUSHA2-1));
      tmp[row][counter] = make_Edge(nonce, tmp[0][0], node0, node1);

      __syncthreads();

      if (counter == FLUSHA-1)
      {
        int localIdx = min(FLUSHA2, counters[row]);
        int newCount = localIdx % FLUSHA;
        int nflush = localIdx - newCount;
        int cnt = min((int)atomicAdd(indexes + row * NX + col, nflush), (int)(maxOut - nflush));

        for (int i = 0; i < nflush; i += TMPPERLL4)
        {
            buffer[((uint64_t)(row * NX + col) * maxOut + cnt + i) / TMPPERLL4] = *(uint4 *)(&tmp[row][i]);
        }

        for (int t = 0; t < newCount; t++)
        {
          tmp[row][t] = tmp[row][t + nflush];
        }

        counters[row] = newCount;
      }
      __syncthreads();
    }

    EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);

    for (int row = lid; row < NX; row += dim)
    {
      int localIdx = min(FLUSHA2, counters[row]);
      for (int j = localIdx; j % TMPPERLL4; j++)
      {
          tmp[row][j] = zero;
      }
      for (int i = 0; i < localIdx; i += TMPPERLL4)
      {
        int cnt = min((int)atomicAdd(indexes + row * NX + col, TMPPERLL4), (int)(maxOut - TMPPERLL4));
        buffer[((uint64_t)(row * NX + col) * maxOut + cnt) / TMPPERLL4] = *(uint4 *)(&tmp[row][i]);
      }
    }
}

template <typename Edge> __device__ bool null(Edge e);

__device__ bool null(uint32_t nonce) {
  return nonce == 0;
}

__device__ bool null(uint2 nodes) {
  return nodes.x == 0 && nodes.y == 0;
}

#ifndef FLUSHB
#define FLUSHB 8
#endif

template<int maxOut, typename EdgeOut>
__global__ void SeedB(const siphash_keys &sipkeys, const EdgeOut * __restrict__ source, uint4 * __restrict__ destination, const int * __restrict__ sourceIndexes, int * __restrict__ destinationIndexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int FLUSHB2 = 2 * FLUSHB;

  __shared__ EdgeOut tmp[NX][FLUSHB2];
  const int TMPPERLL4 = sizeof(uint4) / sizeof(EdgeOut);
  __shared__ int counters[NX];


  for (int col = lid; col < NX; col += dim)
  {
      counters[col] = 0;
  }

  __syncthreads();

  const int row = group / NX;
  const int bucketEdges = min((int)sourceIndexes[group], (int)maxOut);
  const int loops = (bucketEdges + dim-1) / dim;

  for (int loop = 0; loop < loops; loop++)
  {
    int col;
    int counter = 0;
    const int edgeIndex = loop * dim + lid;

    if (edgeIndex < bucketEdges)
    {
      const int index = group * maxOut + edgeIndex;
      EdgeOut edge = __ldg(&source[index]);

      if (null(edge))
      {
          continue;
      }

      uint32_t node1 = endpoint(sipkeys, edge, 0);
      col = (node1 >> XBITS) & XMASK;
      counter = min((int)atomicAdd(counters + col, 1), (int)(FLUSHB2-1));
      tmp[col][counter] = edge;
    }

    __syncthreads();

    if (counter == FLUSHB-1)
    {
      int localIdx = min(FLUSHB2, counters[col]);
      int newCount = localIdx % FLUSHB;
      int nflush = localIdx - newCount;
      int cnt = min((int)atomicAdd(destinationIndexes + row * NX + col, nflush), (int)(maxOut - nflush));

      for (int i = 0; i < nflush; i += TMPPERLL4)
      {
          destination[((uint64_t)(row * NX + col) * maxOut + cnt + i) / TMPPERLL4] = *(uint4 *)(&tmp[col][i]);
      }

      for (int t = 0; t < newCount; t++)
      {
        tmp[col][t] = tmp[col][t + nflush];
      }

      counters[col] = newCount;
    }
    __syncthreads(); 
  }

  EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);

  for (int col = lid; col < NX; col += dim)
  {
    int localIdx = min(FLUSHB2, counters[col]);

    for (int j = localIdx; j % TMPPERLL4; j++)
    {
        tmp[col][j] = zero;
    }

    for (int i = 0; i < localIdx; i += TMPPERLL4)
    {
        int cnt = min((int)atomicAdd(destinationIndexes + row * NX + col, TMPPERLL4), (int)(maxOut - TMPPERLL4));
        destination[((uint64_t)(row * NX + col) * maxOut + cnt) / TMPPERLL4] = *(uint4 *)(&tmp[col][i]);
    }
  }
}

__device__ __forceinline__  void Increase2bCounter(uint32_t *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  uint32_t mask = 1 << bit;

  uint32_t old = atomicOr(ecounters + word, mask) & mask;
  if (old)
    atomicOr(ecounters + word + NZ/32, mask);
}

__device__ __forceinline__  bool Read2bCounter(uint32_t *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  uint32_t mask = 1 << bit;

  return (ecounters[word + NZ/32] & mask) != 0;
}

__device__ uint2 make_Edge(const uint32_t nonce, const uint2 dummy, const uint32_t node0, const uint32_t node1) {
   return make_uint2(node0, node1);
}

__device__ uint2 make_Edge(const uint2 edge, const uint2 dummy, const uint32_t node0, const uint32_t node1) {
   return edge;
}

__device__ uint32_t make_Edge(const uint32_t nonce, const uint32_t dummy, const uint32_t node0, const uint32_t node1)
{
   return nonce;
}

template <typename Edge> uint32_t __device__ endpoint(const siphash_keys &sipkeys, Edge e, int uorv);

__device__ uint32_t endpoint(const siphash_keys &sipkeys, uint32_t nonce, int uorv)
{
  return dipnode(sipkeys, nonce, uorv);
}

__device__ uint32_t endpoint(const siphash_keys &sipkeys, uint2 nodes, int uorv)
{
  return uorv ? nodes.y : nodes.x;
}

template<int maxIn, typename EdgeIn, int maxOut, typename EdgeOut> __global__
void Round(const int round,
           const siphash_keys &sipkeys,
           const EdgeIn * __restrict__ source,
           EdgeOut * __restrict__ destination,
           const int * __restrict__ sourceIndexes,
           int * __restrict__ destinationIndexes)
{
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const static int COUNTERWORDS = NZ / 16; // 16 2-bit counters per 32-bit word

  __shared__ uint32_t ecounters[COUNTERWORDS];

  for (int i = lid; i < COUNTERWORDS; i += dim)
  {
      ecounters[i] = 0;
  }

  __syncthreads();

  const int edgesInBucket = min(sourceIndexes[group], maxIn);
  const int loops = (edgesInBucket + dim-1) / dim;

  for (int loop = 0; loop < loops; loop++)
  {
    const int lindex = loop * dim + lid;

    if (lindex < edgesInBucket)
    {
      const int index = maxIn * group + lindex;
      EdgeIn edge = __ldg(&source[index]);

      if (null(edge))
      {
          continue;
      }

      uint32_t node = endpoint(sipkeys, edge, round&1);
      Increase2bCounter(ecounters, node >> (2*XBITS));
    }
  }
  __syncthreads();

  for (int loop = 0; loop < loops; loop++)
  {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket)
    {
      const int index = maxIn * group + lindex;
      EdgeIn edge = __ldg(&source[index]);

      if (null(edge))
      {
          continue;
      }

      uint32_t node0 = endpoint(sipkeys, edge, round&1);

      if (Read2bCounter(ecounters, node0 >> (2*XBITS)))
      {
        uint32_t node1 = endpoint(sipkeys, edge, (round&1)^1);
        const int bucket = node1 & X2MASK;
        const int bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), maxOut - 1);
        destination[bucket * maxOut + bktIdx] = (round&1) ? make_Edge(edge, *destination, node1, node0) : make_Edge(edge, *destination, node0, node1);
      }
    }
  }
}

template<int maxIn>
__global__ void Tail(const uint2 *source, uint2 *destination, const int *sourceIndexes, int *destinationIndexes) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  int myEdges = sourceIndexes[group];
  __shared__ int destIdx;

  if (lid == 0)
    destIdx = atomicAdd(destinationIndexes, myEdges);
  __syncthreads();
  for (int i = lid; i < myEdges; i += dim)
    destination[destIdx + lid] = source[group * maxIn + lid];
}

#ifdef DEBUG
#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stdout,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
#endif

__global__ void Recovery(const siphash_keys &sipkeys, uint4 *buffer, int *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  const int loops = NEDGES / nthreads;

  __shared__ uint64_t nonces[PROOFSIZE];
  
  if (lid < PROOFSIZE)
  {
      nonces[lid] = 0;
  }

  __syncthreads();

  for (int i = 0; i < loops; i++)
  {
      uint64_t nonce = gid * loops + i;
      uint64_t u = dipnode(sipkeys, nonce, 0);
      uint64_t v = dipnode(sipkeys, nonce, 1);

      for (int i = 0; i < PROOFSIZE; i++)
      {
          if (recoveredges[i].x == u && recoveredges[i].y == v)
          {
              nonces[i] = nonce;
          }
      }
  }
  __syncthreads();

  if (lid < PROOFSIZE)
  {
    if (nonces[lid] > 0)
    {
        indexes[lid] = nonces[lid];
    }
  }

}

struct blockstpb {
  uint16_t blocks;
  uint16_t tpb;
};

struct trimparams {
  uint16_t expand;
  uint16_t ntrims;
  blockstpb genA;
  blockstpb genB;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;

  trimparams() {
    expand              =    0;
    ntrims              =  176;
    genA.blocks         = 4096;
    genA.tpb            =  256;
    genB.blocks         =  NX2;
    genB.tpb            =  128;
    trim.blocks         =  NX2;
    trim.tpb            =  512;
    tail.blocks         =  NX2;
    tail.tpb            = 1024;
    recover.blocks      = 1024;
    recover.tpb         = 1024;
  }
};

typedef uint32_t proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  size_t sizeA, sizeB;
  const size_t indexesSize = NX * NY * sizeof(uint32_t);
  uint4 *bufferA;
  uint4 *bufferB;
  uint4 *bufferAB;
  int *indexesE;
  int *indexesE2;
  uint32_t hostA[NX * NY];
  uint32_t *uvnodes;
  siphash_keys sipkeys, *dipkeys;
  bool abort;
  bool initsuccess = false;


  edgetrimmer(const trimparams _tp) : tp(_tp)
  {
    tp = _tp;
    hipMalloc((void**)&dt, sizeof(edgetrimmer));
    hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(uint64_t));
    hipMalloc((void**)&dipkeys, sizeof(siphash_keys));
    hipMalloc((void**)&indexesE, indexesSize);
    hipMalloc((void**)&indexesE2, indexesSize);

    sizeA = ROW_EDGES_A * NX * (tp.expand > 0 ? sizeof(uint32_t) : sizeof(uint2));
    sizeB = ROW_EDGES_B * NX * (tp.expand > 1 ? sizeof(uint32_t) : sizeof(uint2));
    const size_t bufferSize = sizeA + sizeB;

    hipMalloc((void**)&bufferA, bufferSize);

    bufferB  = bufferA + sizeA / sizeof(uint4);
    bufferAB = bufferA + sizeB / sizeof(uint4);
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
  }

  uint64_t globalbytes() const
  {
    return (sizeA+sizeB) + 2 * indexesSize + sizeof(siphash_keys) + PROOFSIZE * 2 * sizeof(uint32_t) + sizeof(edgetrimmer);
  }

  ~edgetrimmer()
  {
    hipFree(bufferA);
    hipFree(indexesE2);
    hipFree(indexesE);
    hipFree(dipkeys);
    hipFree(uvnodes);
    hipFree(dt);
    hipDeviceReset();
  }

  uint32_t trim()
  {

    hipMemset(indexesE, 0, indexesSize);
    hipMemset(indexesE2, 0, indexesSize);

    hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);
  
    hipDeviceSynchronize();

    if (tp.expand == 0)
    {
        SeedA<EDGES_A, uint2><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, bufferAB, (int *)indexesE);
    }
    else
    {
        SeedA<EDGES_A, uint32_t><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, bufferAB, (int *)indexesE);
    }
    if (abort) return false;
    hipDeviceSynchronize();

    const uint32_t halfA = sizeA/2 / sizeof(uint4);
    const uint32_t halfE = NX2 / 2;

    if (tp.expand == 0)
    {
      SeedB<EDGES_A, uint2><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const uint2 *)bufferAB, bufferA, (const int *)indexesE, indexesE2);
      SeedB<EDGES_A, uint2><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const uint2 *)(bufferAB+halfA), bufferA+halfA, (const int *)(indexesE+halfE), indexesE2+halfE);
    }
    else
    {
      SeedB<EDGES_A, uint32_t><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const uint32_t *)bufferAB, bufferA, (const int *)indexesE, indexesE2);
      SeedB<EDGES_A, uint32_t><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const uint32_t *)(bufferAB+halfA), bufferA+halfA, (const int *)(indexesE+halfE), indexesE2+halfE);
    }

    hipDeviceSynchronize();

    if(will_debug)
    {
        fprintf(stdout,"GPU[%d] Seeding completed\n", global_device_id);
    }

    if (abort) return false;
    hipMemset(indexesE, 0, indexesSize);

    if (tp.expand == 0)
    {
        Round<EDGES_A, uint2, EDGES_B, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632
    }
    else if (tp.expand == 1)
    {
        Round<EDGES_A,   uint32_t, EDGES_B, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const uint32_t *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632
    }
    else // tp.expand == 2
    {
        Round<EDGES_A,   uint32_t, EDGES_B,   uint32_t><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const uint32_t *)bufferA, (  uint32_t *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632
    }
    if (abort) return false;

    hipMemset(indexesE2, 0, indexesSize);

    if (tp.expand < 2)
    {
        Round<EDGES_B, uint2, EDGES_B/2, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(1, *dipkeys, (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .296
    }
    else
    {
        Round<EDGES_B,   uint32_t, EDGES_B/2, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(1, *dipkeys, (const uint32_t *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .296
    }

    if (abort) return false;
    hipMemset(indexesE, 0, indexesSize);
    Round<EDGES_B/2, uint2, EDGES_A/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(2, *dipkeys, (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .176
    if (abort) return false;
    hipMemset(indexesE2, 0, indexesSize);
    Round<EDGES_A/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(3, *dipkeys, (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .117
  
    hipDeviceSynchronize();
  
    for (int round = 4; round < tp.ntrims; round += 2)
    {
        if (abort) return false;
        hipMemset(indexesE, 0, indexesSize);
        Round<EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(round, *dipkeys,  (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE);
        if (abort) return false;
        hipMemset(indexesE2, 0, indexesSize);
        Round<EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(round+1, *dipkeys,  (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2);
    }
    
    if (abort) return false;
    hipMemset(indexesE, 0, indexesSize);
    hipDeviceSynchronize();
  
    Tail<EDGES_B/4><<<tp.tail.blocks, tp.tail.tpb>>>((const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE);
    hipMemcpy(hostA, indexesE, NX * NY * sizeof(uint32_t), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    return hostA[0];
  }

};

#define IDXSHIFT 10
#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash
{
    public:
        uint64_t *cuckoo;

        cuckoo_hash()
        {
          cuckoo = new uint64_t[CUCKOO_SIZE];
        }

        ~cuckoo_hash()
        {
          delete[] cuckoo;
        }

        void set(node_t u, node_t v)
        {
            uint64_t niew = (uint64_t)u << NODEBITS | v;
            for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK)
            {
                uint64_t old = cuckoo[ui];
                if (old == 0 || (old >> NODEBITS) == (u & KEYMASK))
                {
                    cuckoo[ui] = niew;
                    return;
                }
            }
        }

        node_t operator[](node_t u) const
        {
            for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK)
            {
                uint64_t cu = cuckoo[ui];

                if (!cu)
                {
                    return 0;
                }

                if ((cu >> NODEBITS) == (u & KEYMASK))
                {
                  return (node_t)(cu & NODEMASK);
                }
            }
        }
};

const static uint32_t MAXPATHLEN = 8 << ((NODEBITS+2)/3);

int nonce_cmp(const void *a, const void *b)
{
    return *(uint32_t *)a - *(uint32_t *)b;
}

const static uint32_t MAXEDGES = 0x20000;

struct solver_ctx
{
    edgetrimmer trimmer;
    bool mutatenonce;
    uint2 *edges;
    cuckoo_hash *cuckoo;
    uint2 soledges[PROOFSIZE];

    std::vector<uint32_t> sols; // concatenation of all proof's indices

    uint32_t us[MAXPATHLEN];
    uint32_t vs[MAXPATHLEN];

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp)
  {
        edges   = new uint2[MAXEDGES];
        cuckoo  = new cuckoo_hash();
        mutatenonce = mutate_nonce;
   }

  void setheadernonce(char * const headernonce, const uint32_t len, const uint64_t nonce) {
    if (mutatenonce) {
      // The KeyHash takes 44 byte - put nonce at 45-56
      base64_encode_nonce(nonce, headernonce + 44);
    }
    setheader(headernonce, len, &trimmer.sipkeys);
        sols.clear();
    }

    ~solver_ctx()
    {
      delete cuckoo;
      delete[] edges;
    }

    void recordedge(const uint32_t i, const uint32_t u2, const uint32_t v2)
    {
        soledges[i].x = u2/2;
        soledges[i].y = v2/2;
    }

    void solution(const uint32_t *us, uint32_t nu, const uint32_t *vs, uint32_t nv)
    {
      uint32_t ni = 0;
      recordedge(ni++, *us, *vs);

      while (nu--)
      {
          recordedge(ni++, us[(nu+1)&~1], us[nu|1]); // u's in even position; v's in odd
      }
      while (nv--)
      {
          recordedge(ni++, vs[nv|1], vs[(nv+1)&~1]); // u's in odd position; v's in even
      }

      sols.resize(sols.size() + PROOFSIZE);

      hipMemcpyToSymbol(HIP_SYMBOL(recoveredges), soledges, sizeof(soledges));
    hipMemset(trimmer.indexesE2, 0, trimmer.indexesSize);
    Recovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>(*trimmer.dipkeys, trimmer.bufferA, (int *)trimmer.indexesE2);
    hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer.indexesE2, PROOFSIZE * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

      qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(uint32_t), nonce_cmp);
    }

    uint32_t path(uint32_t u, uint32_t *us)
    {
      uint32_t nu, u0 = u;
      for (nu = 0; u; u = (*cuckoo)[u])
      {
        if (nu >= MAXPATHLEN)
        {
          while (nu-- && us[nu] != u) ;
          if (~nu)
          {
            fprintf(stdout, "illegal %4d-cycle from node %d\n", MAXPATHLEN-nu, u0);
            exit(0);
          }

          fprintf(stdout, "maximum path length exceeded\n");
          return 0; // happens once in a million runs or so; signal trouble
        }

        us[nu++] = u;
      }

      return nu;
    }

    void addedge(uint2 edge)
    {
      const uint32_t u0 = edge.x << 1, v0 = (edge.y << 1) | 1;
      if (u0)
      {
        uint32_t nu = path(u0, us), nv = path(v0, vs);
        if (!nu-- || !nv--)
        {
            return; // drop edge causing trouble
        }

        if (us[nu] == vs[nv])
        {
          const uint32_t min = nu < nv ? nu : nv;
          for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;

          const uint32_t len = nu + nv + 1;

          if(will_debug)
          {
              fprintf(stdout, "GPU[%d] %4d-cycle found\n", global_device_id, len);
          }

          if (len == PROOFSIZE)
          {
             solution(us, nu, vs, nv);
          }

        }
        else if (nu < nv)
        {
          while (nu--)
          {
              cuckoo->set(us[nu+1], us[nu]);
          }
          cuckoo->set(u0, v0);
        }
        else
        {
          while (nv--)
          {
             cuckoo->set(vs[nv+1], vs[nv]);
          }

          cuckoo->set(v0, u0);
        }
      }
    }

    void findcycles(uint2 *edges, uint32_t nedges)
    {
      memset(cuckoo->cuckoo, 0, CUCKOO_SIZE * sizeof(uint64_t));

      for (uint32_t i = 0; i < nedges; i++)
      {
          addedge(edges[i]);
      }

    }

    int solve()
    {
      uint32_t timems,timems2;
      auto time0 = std::chrono::high_resolution_clock::now();
      trimmer.abort = false;
      uint32_t nedges = trimmer.trim();
      if (!nedges)
      {
          return 0;
      }

      if (nedges > MAXEDGES)
      {
        fprintf(stdout, "OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
        nedges = MAXEDGES;
      }

      hipMemcpy(edges, trimmer.bufferB, nedges * 8, hipMemcpyDeviceToHost);

      auto time1 = std::chrono::high_resolution_clock::now();
      auto duration = std::chrono::duration_cast<ms>(time1 - time0);

      timems = duration.count();
      time0 = std::chrono::high_resolution_clock::now();
      findcycles(edges, nedges);
      time1 = std::chrono::high_resolution_clock::now();
      duration = std::chrono::duration_cast<ms>(time1 - time0);
      timems2 = duration.count();
      
      if(will_debug)
      {
         fprintf(stdout, "GPU[%d] findcycles edges %d time %d ms total %d ms\n", global_device_id, nedges, timems2, timems+timems2);
      }
      
      return sols.size() / PROOFSIZE;
    }

    void abort()
    {
        trimmer.abort = true;
    }

};

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80
typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               uint64_t nonce,
                               uint32_t range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  uint64_t time0, time1;
  uint32_t timems;
  SolverParams params;

  uint64_t time_all_start,time_all_end;
  uint32_t time_ms_all;

  uint32_t sumnsols = 0;
  int device_id;
  char my_solution[1024];

  time_all_start = timestamp();
  if(will_debug)
  {
    if (stats != NULL)
    {
      hipGetDevice(&device_id);
      hipDeviceProp_t props;
      hipGetDeviceProperties(&props, stats->device_id);
      stats->device_id = device_id;
      stats->edge_bits = EDGEBITS;
      strncpy(stats->device_name, props.name, MAX_NAME_LEN);
    }

    if (ctx == NULL || !ctx->trimmer.initsuccess)
    {
        print_log("Error initialising trimmer. Aborting.\n");
        print_log("Reason: %s\n", LAST_ERROR_REASON);
        if (stats != NULL)
        {
           stats->has_errored = true;
           strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
        }
        return 0;
    }
  }

  uint32_t nsols = 0;
  will_debug = true;
  
  for (uint32_t r = 0; r < range; r++)
  {
    if(will_debug)
    {
        time0 = timestamp();
        ctx->setheadernonce(header, header_length, nonce + r);
        print_log("GPU[%d] nonce %llu k0 k1 k2 k3 %llx %llx %llx %llx\n",global_device_id, nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
        nsols = ctx->solve();
        time1 = timestamp();
        timems = (time1 - time0) / 1000000;
        print_log("GPU[%d] Time: %d ms\n",global_device_id, timems);
        if (timems == 0)
        {
            print_log("GPU[%d] We stop and retry because time to low: is %d ms\n",global_device_id, timems); exit(-1);
        }
    }
    else
    {
        time0 = timestamp();
        ctx->setheadernonce(header, header_length, nonce + r);
        nsols = ctx->solve();
        time1 = timestamp();
        timems = (time1 - time0) / 1000000;
        if (timems == 0)
        {
            print_log("GPU[%d] We stop and retry because time to low: is %d ms\n",global_device_id, timems); exit(-1);
        }
    }

    char temps[512];
    temps[0]=0;

    for (unsigned s = 0; s < nsols; s++)
    {
      sprintf(temps,"(%jx)", (uintmax_t)(nonce+r));
      strcat(my_solution, temps);
      uint32_t* prf = &ctx->sols[s * PROOFSIZE];

       for (uint32_t i = 0; i < PROOFSIZE; i++)
       {
           temps[0]=0;
           sprintf(temps," %jx",(uintmax_t)prf[i]);
           strcat(my_solution, temps);
       }
       
       fprintf(stdout,"Solution%s\n",my_solution);
       my_solution[0]=0;

      if (solutions != NULL)
      {
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;

        for (uint32_t i = 0; i < PROOFSIZE; i++)
        {
            solutions->sols[sumnsols+s].proof[i] = (uint64_t) prf[i];
        }
      }

      if(will_debug)
      {
          int pow_rc = verify(prf, &ctx->trimmer.sipkeys);
          if (pow_rc == POW_OK)
          {
            print_log("GPU[%d] Verified with cyclehash ",global_device_id);
            unsigned char cyclehash[32];
            blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);

            for (int i=0; i<32; i++)
            {
                print_log("%02x", cyclehash[i]);
            }
              print_log("\n");
          }
          else
          {
              print_log("GPU[%d] FAILED due to %s\n", global_device_id, errstr[pow_rc]);
          }
      }

    }

    sumnsols += nsols;

    if(will_debug)
    {
        if (stats != NULL)
        {
            stats->last_start_time = time0;
            stats->last_end_time = time1;
            stats->last_solution_time = time1 - time0;
        }
    }

  } // end for loop

  time_all_end = timestamp();
  time_ms_all = (time_all_end - time_all_start) / 1000000;

  print_log("GPU[%d] %d total solutions of %d nonces Time: %d ms\n",global_device_id, sumnsols, range, time_ms_all);
  
  return sumnsols > 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
  tp.expand = params->expand;
  tp.genA.blocks = params->genablocks;
  tp.genA.tpb = params->genatpb;
  tp.genB.tpb = params->genbtpb;
  tp.trim.tpb = params->trimtpb;
  tp.tail.tpb = params->tailtpb;
  tp.recover.blocks = params->recoverblocks;
  tp.recover.tpb = params->recovertpb;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, params->device);

  if(will_debug)
  {
    assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
    assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
    assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
    // assert(tp.tailblocks <= prop.threadDims[0]);
    assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
    assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

    assert(tp.genA.blocks * tp.genA.tpb <= NEDGES); // check THREADS_HAVE_EDGES
    assert(tp.recover.blocks * tp.recover.tpb <= NEDGES); // check THREADS_HAVE_EDGES
    assert(tp.genA.tpb / NX <= FLUSHA); // check ROWS_LIMIT_LOSSES
    assert(tp.genA.tpb / NX <= FLUSHA); // check COLS_LIMIT_LOSSES
  }


  hipSetDevice(params->device);

  if (!params->cpuload)
  {
      hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  }

  SolverCtx* ctx = new SolverCtx(tp, params->mutate_nonce);

  return ctx;
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params)
{
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->expand = tp.expand;
  params->genablocks = tp.genA.blocks;
  params->genatpb = tp.genA.tpb;
  params->genbtpb = tp.genB.tpb;
  params->trimtpb = tp.trim.tpb;
  params->tailtpb = tp.tail.tpb;
  params->recoverblocks = tp.recover.blocks;
  params->recovertpb = tp.recover.tpb;
  params->cpuload = true;
}



int main(int argc, char **argv)
{
    trimparams tp;

    char bversion[256];
    sprintf(bversion,"%s", __BUILD_VERSION);

    char bdate[256];
    sprintf(bdate,"%s", __BUILD_DATE);

    char buildby[256];
        sprintf(buildby,"%s", __BUILD_BY);

    char build_sha[256];
    sprintf(build_sha,"%s", __BUILD_SHA);

    uint64_t nonce = 0;
    uint32_t range = 1;
    uint32_t device = 0;
    char header[HEADERLEN];
    uint32_t len;
    int opt;
    bool cpuload = false;
    //FILE *my_logfile;
    //my_logfile = fopen("solution.log", "w");

  // set defaults
  SolverParams params;
  fill_default_params(&params);

    memset(header, 0, sizeof(header));
    static const char *optString = "scb:c:d:E:h:k:m:n:r:U:u:v:w:y:Z:z:gb:";

    while ((opt = getopt(argc, argv, optString)) != -1)
    {
        switch (opt)
        {
           case 'c':
               cpuload = true;
               break;
           case 's':
               fprintf(stdout, "SYNOPSIS\n  %s \n[-d device] \n[-E 0-2] \n[-h hexheader] \n[-m trims] \n[-n nonce] \n[-r range] \n[-U seedAblocks] \n[-u seedAthreads] \n[-v seedBthreads] \n[-w Trimthreads] \n[-y Tailthreads] \n[-Z recoverblocks] \n[-z recoverthreads] \n[-g debug] \n[-c cpu none blocking]\n", argv[0]);
               fprintf(stdout, "\n DEFAULTS\n  %s -d %d -E %d -h \"\" -m %d -n %zd -r %d -U %d -u %d -v %d -w %d -y %d -Z %d -z %d\n", argv[0], device, tp.expand, tp.ntrims, nonce, range, tp.genA.blocks, tp.genA.tpb, tp.genB.tpb, tp.trim.tpb, tp.tail.tpb, tp.recover.blocks, tp.recover.tpb);
               fprintf(stdout, "Build version  : %s by %s source_sha256: %s\n", bversion, buildby,build_sha);
               fprintf(stdout, "Build date: %s\n", bdate);
               exit(0);
           case 'd':
               params.device = atoi(optarg);
               global_device_id = params.device;
               break;
           case 'E':
               params.expand = atoi(optarg);
               break;
           case 'g':
                will_debug = true;
                break;
           case 'h':
               len = strlen(optarg)/2;
               for (uint32_t i=0; i<len; i++)
               {
                   sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
               }
               break;
           case 'n':
               nonce = strtoull(optarg, NULL, 10);
               break;
           case 'm':
               params.ntrims = atoi(optarg) & -2; // make even as required by solve()
               break;
           case 'r':
               range = atoi(optarg);
               break;
           case 'U':
               params.genablocks = atoi(optarg); // genA.blocks
               break;
           case 'u':
               params.genatpb = atoi(optarg); // genA.tpb
               break;
           case 'v':
               params.genbtpb = atoi(optarg);
               break;
           case 'w':
               params.trimtpb = atoi(optarg);
               break;
           case 'y':
               params.tailtpb = atoi(optarg);
               break;
           case 'Z':
               params.recoverblocks = atoi(optarg);
               break;
           case 'z':
               params.recovertpb = atoi(optarg);
               break;
      }
    }

    //will_debug = true;

    int nDevices;
    hipGetDeviceCount(&nDevices);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, global_device_id);

    hipSetDevice(global_device_id);
    
    if (cpuload)
    {
        // may be for old and pure systems but not for highspeed machines!!
        hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    }
    else
    {
        // best performance
        hipSetDeviceFlags(hipDeviceScheduleYield);
    }

    if (will_debug)
    {
        uint64_t dbytes = prop.totalGlobalMem;
        int dunit;
        for (dunit=0; dbytes >= 10240; dbytes>>=10,dunit++) ;

        fprintf(stdout, "GPU[%d] %s with %d%cB @ %d bits x %dMHz\n", global_device_id, prop.name, (uint32_t)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);

        fprintf(stdout, "GPU[%d] Looking for %d-cycle on cuckoo%d(\"%s\",%zd", global_device_id,PROOFSIZE, NODEBITS, header, nonce);

        if (range > 1)
        {
            fprintf(stdout, "-%zd", nonce+range-1);
        }
        fprintf(stdout, ") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, tp.ntrims, NX);
    }

    SolverCtx* ctx = create_solver_ctx(&params);
    //solver_ctx ctx(tp);

    if (will_debug)
    {
        uint64_t bytes = ctx->trimmer.globalbytes();
        int unit;

        for (unit=0; bytes >= 10240; bytes>>=10,unit++) ;

        fprintf(stdout, "Using %d%cB of global memory.\n", (uint32_t)bytes, " KMGT"[unit]);

    }

    run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

    return 0;

}
